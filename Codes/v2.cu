#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <time.h>
#include <string.h>

#define PI 3.1415926536
#define e 2.718281828459

#define N 64*64
#define PATCH 3
#define RADIUS (PATCH-1)/2
#define THREADS_PER_BLOCK 64

struct timeval tic(){
    struct timeval tv;
    gettimeofday(&tv,NULL);
    return tv;
}

typedef struct Patches
{
    int index;
    float central;
    float* patchArray;
}Patch;

double toc(struct timeval begin){
    struct timeval end;
    gettimeofday(&end,NULL);
    double stime = ((double)(end.tv_sec-begin.tv_sec)*1000)+((double)(end.tv_usec-begin.tv_usec)/1000);
    stime = stime / 1000;
    return (stime);
}

float* readFile(int n, int m, char *file_path){
    FILE* ptrFile = fopen(file_path, "r");

    float *I = (float*)malloc(n*m*sizeof(float));

    if (!ptrFile){
        printf("Error Reading File\n");
        exit (0);
    }

    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){
            fscanf(ptrFile,"%f,", &I[n*i+j]);
        }
    }

    fclose(ptrFile);

    return I;
}

void toTXT(float* array,char *output, int n, int m){
    FILE *fp;

    fp=fopen(output,"w");

    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){
            if(j<m-1){
                fprintf(fp,"%lf,",array[n*i+j]);
            }else if(j==m-1){
                fprintf(fp,"%lf",array[n*i+j]);
            }
        }
        fprintf(fp,"\n",array[n*i]);
    }
    fclose(fp);
    printf("File %s saved.\n", output);
}

__global__ void normalization(float* A, float* B, float min, float max){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N)
        B[i] = (A[i] - min) / max;
}

float AWGN_generator()     //https://www.embeddedrelated.com/showcode/311.php
{/* Generates additive white Gaussian Noise samples with zero mean and a standard deviation of 1. */
  float dev = 0.03162; //var = 0.01
  float temp1;
  float temp2;
  float result;
  int p = 1;

  while( p > 0 )
  {
	temp2 = ( rand() / ( (float)RAND_MAX ) ); /*  rand() function generates an
                                                       integer between 0 and  RAND_MAX,
                                                       which is defined in stdlib.h.
                                                   */

    if ( temp2 == 0 )
    {// temp2 is >= (RAND_MAX / 2)
        p = 1;
    }// end if
    else
    {// temp2 is < (RAND_MAX / 2)
        p = -1;
    }// end else

  }// end while()

  temp1 = cos( ( 2.0 * (float)PI ) * rand() / ( (float)RAND_MAX ) );
  result = sqrt( -2.0 * log( temp2 ) ) * temp1;

  return result * dev;	// return the generated random sample to the caller

}// end AWGN_generator()

Patch* makePatches(float* J, int n, int m, Patch* allPatches, int patchSizeH, int patchSizeW){
    int mdW = (patchSizeW - 1)/2;
    int mdH = (patchSizeH - 1)/2;
    
    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){

            for(int w=0; w<patchSizeW; w++){
                for(int h=0; h<patchSizeH; h++){
                    allPatches[n*j+i].patchArray[patchSizeH*w+h] = 0;
                }
            }

            allPatches[n*j+i].central = J[n*j+i];
            allPatches[n*j+i].index = n*j+i;

            if(i==0 && j==0){
                for(int w=mdW; w<patchSizeW; w++){
                    for(int h=mdH; h<patchSizeH; h++){
                        allPatches[n*j+i].patchArray[patchSizeH*w+h] = J[(n*j+i)-(mdW-w)*n-(mdH-h)];
                    }
                } 
            }else if(i>0 && j==0){
                for(int h=0; h<patchSizeH-1; h++){
                    for(int w=0; w<patchSizeW; w++){
                        allPatches[n*j+i].patchArray[patchSizeH*w+h] = allPatches[n*j+(i-1)].patchArray[patchSizeH*w+(h+1)];
                    }
                }
               

                for(int w=mdW; w<patchSizeW; w++){
                    if((n-1-i) >= mdH){
                        allPatches[n*j+i].patchArray[patchSizeH*w+(patchSizeH-1)] = J[(n*j+i)-(mdW-w)*n+mdH];
                    }else if((n-1-i) < mdH){
                        allPatches[n*j+i].patchArray[patchSizeH*w+(patchSizeH-1)] = 0;
                    }
                }
            }else if(j>0){
                for(int w=0; w<patchSizeW-1; w++){
                    for(int h=0; h<patchSizeH; h++){
                        allPatches[n*j+i].patchArray[patchSizeH*w+h] = allPatches[n*(j-1)+i].patchArray[patchSizeH*(w+1)+h];
                    }
                }

                int a,b;
                if(i>=mdH && (n-1-i)>=mdH){
                    a = 0;
                    b = patchSizeH;
                }else if(i<mdH && (n-1-i)>=mdH){
                    a = mdH - i;
                    b = patchSizeH;
                }else if(i<mdH && (n-1-i)<mdH){
                    a = mdH - i;
                    b = mdH + (n-i);
                }else if(i>=mdH && (n-1-i)<mdH){
                    a = 0;
                    b = mdH + (n-i);
                }

                for(int h=a; h<b; h++){
                    if((m-1-j) >= mdW){
                        allPatches[n*j+i].patchArray[patchSizeH*(patchSizeW-1)+h] = J[(n*j+i)+mdW*n-(mdH-h)];
                    }else if((m-1-j) < mdW){
                        allPatches[n*j+i].patchArray[patchSizeH*(patchSizeW-1)+h] = 0;
                    }
                }
            }
        }
    }

    return allPatches;
}


float* computeG_a(int patchSizeH, int patchSizeW, float patchSigma){
    float* gauss = (float*)malloc(patchSizeH*patchSizeW*sizeof(float));
    float max = -1.0;

    for (int i = 0; i < patchSizeH; i++) {
        for (int j = 0; j < patchSizeW; j++) {
            float y = i - (patchSizeH - 1) / 2.0;
            float x = j - (patchSizeW - 1) / 2.0;
            gauss[patchSizeW*i+j] = (1/2.0) * exp(-(x * x + y * y) / (2.0 * PI * patchSigma * patchSigma));
        }
    }    
    return gauss;
}

__global__ void dist(float *W,float *p_i, int i, float *A, float *V, int n, int patchSizeH, float filtSigma){
    float d=0;
    int sizeofRow = n + 2*RADIUS;

    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int index = threadIdx.x;

    __shared__ float d_o;
    __shared__ float sh_gauss[PATCH*PATCH];
    extern __shared__ float sh_A[];

    for(int v=0; v<patchSizeH*patchSizeH; v++){
        sh_gauss[v] = V[v];
    }    

    int x = k/n + RADIUS;
    int y = k%n + RADIUS;
    int indexX = index/n; 
    int indexY = index%n;

    if(k<N){
        sh_A[sizeofRow*(RADIUS+indexX) + (RADIUS+indexY)] = A[sizeofRow*x+y];
        if(indexX<RADIUS && indexY<RADIUS){
            sh_A[sizeofRow*indexX + indexY] = A[sizeofRow*(x-RADIUS) + (y-RADIUS)];
            sh_A[sizeofRow*(indexX+RADIUS+1) + (RADIUS+n+ indexY)] = A[sizeofRow*(x+n) + (y+n)];
        }else if(indexY<RADIUS){
            sh_A[sizeofRow*(indexX+RADIUS) + indexY] = A[sizeofRow*x + (y-RADIUS)];
            sh_A[sizeofRow*indexX + (indexY+n)] = A[sizeofRow*x + (y+n)];
        }else if(indexX<RADIUS){
            sh_A[sizeofRow*indexX + (indexY+RADIUS)] = A[sizeofRow*(x-RADIUS) + y];
            sh_A[sizeofRow*(indexX+RADIUS+1) + (indexY+RADIUS)] = A[sizeofRow*(x+n) + y];
        }
    }
    __syncthreads();


    if(i/THREADS_PER_BLOCK == blockIdx.x){
        int thr = i%THREADS_PER_BLOCK;
        //the coordinates of i in the block
        int x = thr/n;
        int y = thr%n;

        for (int r = 0; r < patchSizeH; r++) {
            for(int c=0; c<patchSizeH; c++){
                d += sh_gauss[patchSizeH*r+c] * powf(sh_A[(x*n+y)+n*r+c] - sh_A[(indexX*n+indexY)+n*r+c],2);
            }
        }
    }else{
        for (int r = 0; r < patchSizeH; r++) {
            for(int c=0; c<patchSizeH; c++){
                d += sh_gauss[patchSizeH*r+c] * powf(p_i[n*r+c] - sh_A[(indexX*n+indexY)+n*r+c],2);
            }
        }

    }
    
    d = sqrt(d);
    W[k] = exp(-pow(d,2) / filtSigma);
    d=0;
      
}

__global__ void dim(float *w, float *z){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i<N){
        w[i] = w[i] / *z;
    }
}

int main(int argc, char *argv[]){
    int n = atoi(argv[1]);
    int m = atoi(argv[2]);
    int patchSizeH = atoi(argv[3]);
    int patchSizeW = atoi(argv[4]);
   
    float patchSigma =5/3;
    float filtSigma =0.01 ;

    char* file_path;
    file_path=(char*)malloc(strlen(argv[5])*sizeof(char));
    memcpy(file_path,argv[5],strlen(argv[5]));

    int size = N * sizeof(float);
    int sizePatch = patchSizeH * patchSizeW * sizeof(float);
    int pSize = patchSizeH * patchSizeW;
    int s = n+(patchSizeH-1);
    int sA = s*s; 

    float *I, *I_norm, *J, *If;
    float *dev_I, *dev_I_norm, *dev_J, *dev_gauss;
    float *P, *w;
    float *A = (float*)malloc(sA*sizeof(float)); 
   
    //allocate memory for device copies
    hipMalloc(&dev_I, size);
    hipMalloc(&dev_I_norm, size);
    hipMalloc(&dev_J, size);
    hipMalloc(&dev_gauss, sizePatch);    

    I = (float*)malloc(size);
    I_norm = (float*)malloc(size);
    J = (float*)malloc(size);
    If = (float*)malloc(size);
    
    Patch* allPatches;
    allPatches = (Patch*)malloc(n*m*sizeof(Patch));

    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){
            allPatches[n*j+i].patchArray = (float*)malloc(patchSizeH*patchSizeW*sizeof(float));
        }
    }    

    w = (float*)malloc(N*N*sizeof(float));
    float* gauss = (float*)malloc(sizePatch);
    float* Z = (float*)malloc(size);

    struct timeval tStart;

    I = readFile(n,m,file_path);

    //find min of 'I' and max of 'I-min'
    float min = INFINITY;
    float max = -1.0;

    for(int i=0; i<n; i++){
        for(int j=0; j<m; j++){
            if(I[n*i+j]<min) min= I[n*i+j];
        }
    }    

    for(int i=0; i<n*m; i++){
        if((I[i]-min)>max) max = I[i]-min;
    }

    hipMemcpy(dev_I, I, size, hipMemcpyHostToDevice);
    normalization<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_I, dev_I_norm, min, max);
    hipMemcpy(I_norm, dev_I_norm, size, hipMemcpyDeviceToHost);

    for(int i=0; i<n*m; i++){
        J[i] = I_norm[i] + AWGN_generator();
    }
    
    toTXT(I_norm,"normShared.txt",n,m);
    toTXT(J,"JShared.txt",n,m);

   // A : extended J array with zeros all around  
    for(int i=0; i<s; i++){
        for(int j=0; j<(patchSizeH-1)/2; j++){
            A[s*j+i] = 0;
        }
        for(int j=0; j<(patchSizeH-1)/2; j++){
            A[s*(n+(patchSizeH-1)/2)*j+i] = 0;
        }
        for(int j=0; j<(patchSizeH-1)/2; j++){
            A[((patchSizeH-1)/2)*i+j] = 0;
        }
        for(int j=n+(patchSizeH-1)/2; j<s;j++){
            A[((patchSizeH-1)/2)*i+j] = 0;
        }
    }

    for(int i=0; i<n; i++){
        for(int j=0; j<n; j++){
            A[((patchSizeH-1)/2)*s+s*i+(patchSizeH-1)/2+j] = J[n*i+j];
        }
    }    

    allPatches = makePatches(J,n,m,allPatches,patchSizeH,patchSizeW);

    P = (float*)malloc(N*pSize*sizeof(float));
    for(int i=0; i<N; i++){
        for(int j=0; j<pSize; j++){
            P[pSize*i+j] = allPatches[i].patchArray[j];
        }
    }

    float *dev_A;
    hipMalloc(&dev_A, sA*sizeof(float));
    hipMemcpy(dev_A, A, sA*sizeof(float), hipMemcpyHostToDevice);    
    gauss = computeG_a(patchSizeH, patchSizeW, patchSigma);
    hipMemcpy(dev_gauss, gauss, sizePatch, hipMemcpyHostToDevice);

    float *patch_i = (float*)malloc(sizePatch);
    float *dev_patchI;
    hipMalloc(&dev_patchI, sizePatch);
    float *wi_j = (float*)malloc(N*sizeof(float));
    float *dev_wij;
    hipMalloc(&dev_wij, N*sizeof(float));

    tStart = tic();

    for(int i=0; i<N; i++){ 
        for(int j=0; j<pSize; j++){ 
            patch_i[j] = P[pSize*i +j];
        }
        hipMemcpy(dev_patchI, patch_i, sizePatch, hipMemcpyHostToDevice);
        size_t size_shared = s*patchSizeH*sizeof(float);
        dist<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK,size_shared>>>(dev_wij, dev_patchI,i,dev_A, dev_gauss,n,patchSizeH,filtSigma);
        hipMemcpy(wi_j, dev_wij, size, hipMemcpyDeviceToHost);
        for(int j=0; j<N;j++){
            Z[i] += wi_j[j]; 
            w[N*i+j] = wi_j[j];
        }
    }
    double time = toc(tStart);
    
    float *dev_Z;
    hipMalloc(&dev_Z, sizeof(float));
    for(int i=0; i<N; i++){
        for(int j=0; j<N;j++){
            wi_j[j] = w[N*i+j];
        }
        hipMemcpy(dev_wij, wi_j,size, hipMemcpyHostToDevice);
        hipMemcpy(dev_Z, &Z[i], sizeof(float), hipMemcpyHostToDevice);
        dim<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(dev_wij,dev_Z);
        hipMemcpy(wi_j, dev_wij, size,hipMemcpyDeviceToHost);
        for(int j=0; j<N;j++){
            w[N*i+j] = wi_j[j];
            If[i] += w[N*i+j] * J[j];
        }
    }

    toTXT(If,"IfShared.txt",n,m);

    // float *x = (float*)malloc(N*sizeof(float));
    // for(int i=0; i<N; i++){
    //     for(int j=0; j<N; j++){
    //         x[i] += w[N*i+j]; 
    //     }
    // }
    // for(int i=0; i<50; i++){
    //     printf("%f ", x[i]);
    // }

    float* Dif = (float*)malloc(N*sizeof(float));
    for(int i=0; i<N; i++){
        Dif[i] =If[i] - J[i] ;
    }
    toTXT(Dif,"DifShared.txt",n,m);

    printf("Time: %f sec", time);
    
    hipFree(dev_I); hipFree(dev_I_norm); hipFree(dev_J); hipFree(dev_gauss);
    hipFree(dev_patchI); hipFree(dev_wij); hipFree(dev_A); 
    free(I); free(I_norm); free(J); free(patch_i); free(gauss); free(wi_j); free(Z); free(If); free(A);

    return 0;
}